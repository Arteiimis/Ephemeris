#include "hip/hip_runtime.h"
#define INLINE __device__
#include"ephemeris.h"
#include<stdlib.h>
#include<algorithm>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>

#define WARP_SIZE 32
#define CUDA_CORES 1920
#define MAXBLOCKS (1920/32)

#define cuda_max(a,b) ((a)>(b)?(a):(b))

//geopotential data, mlist[first].gpmodel==second
typedef std::vector<std::pair<int_t,geopotential*>> gpdata_t;
//ring data, mlist[first].ringmodel==second
typedef std::vector<std::pair<int_t,ring*>> ringdata_t;

#if 0
#define mycudaMalloc hipMalloc
#define mycudaFree hipFree
#define mymalloc malloc
#define myfree myfree
#else
void mycudaMalloc(void *devPtr,size_t size){
    static void *mem=0;
    static size_t memsize=0;
    if(memsize<size){
        hipFree(mem);
        hipMalloc(&mem,size);
        memsize=size;
    }
    *(void**)devPtr=mem;
}
void *mymalloc(size_t size){
    static void *mem=0;
    static size_t memsize=0;
    if(memsize<size){
        free(mem);
        mem=malloc(size);
        memsize=size;
    }
    return mem;
}
void myfree(void *){

}
void mycudaFree(void *){

}
#endif

struct cuda_rungekutta_kernel_config{
    //number of mass
    int nmass;
    int nblocks,mass_per_block;
    int nthreads,mass_per_thread;
    int_t n_step;
    fast_real dt;
    mass *dmlist;
    mass_state *x0,*f;
    real t_eph;

    void load(std::vector<mass> &mlist,gpdata_t &mgp,ringdata_t &mrg,fast_real _dt,int_t _nstep){
        dt=_dt;
        n_step=_nstep;
        int mn=mlist.size();
        nmass=mn;
        mass_per_block=(mn+MAXBLOCKS-1)/MAXBLOCKS;
        nblocks=(mn+mass_per_block-1)/mass_per_block;
        nthreads=WARP_SIZE*std::min(MAXBLOCKS/nblocks,(mn+WARP_SIZE-1)/WARP_SIZE);
        //Make sure nthreads is power of 2
        int new_nth;
        while(new_nth=nthreads&nthreads-1)nthreads=new_nth;
        mass_per_thread=(mn+nthreads-1)/nthreads;
        int_t grsize=0;
        for(int_t i=0;i<mn;++i){
            mass &m=mlist[i];
            if(m.gpmodel){
                int_t thissize=m.gpmodel->size();
                mgp.push_back({i,m.gpmodel});
                m.gpmodel=(geopotential*)grsize;
                grsize+=thissize;
            }
            if(m.ringmodel){
                int_t thissize=m.ringmodel->size();
                mrg.push_back({i,m.ringmodel});
                m.ringmodel=(ring*)grsize;
                grsize+=thissize;
            }
        }
        mycudaMalloc(&x0,nmass*(sizeof(mass)+26*sizeof(mass_state))+grsize);
        f=x0+nmass;
        dmlist=(mass*)(x0+26*nmass);
        void *grdata=mymalloc(grsize);
        for(auto &mgpi:mgp){
            int_t gpoffset=(int_t)mlist[mgpi.first].gpmodel;
            mlist[mgpi.first].gpmodel=(geopotential*)(gpoffset+(int_t)(dmlist+nmass));
            memcpy((geopotential*)(gpoffset+(int_t)grdata),mgpi.second,mgpi.second->size());
        }
        for(auto &mrgi:mrg){
            int_t rgoffset=(int_t)mlist[mrgi.first].ringmodel;
            mlist[mrgi.first].ringmodel=(ring*)(rgoffset+(int_t)(dmlist+nmass));
            memcpy((ring*)(rgoffset+(int_t)grdata),mrgi.second,mrgi.second->size());
        }
        hipMemcpy(dmlist,mlist.data(),nmass*sizeof(mass),hipMemcpyHostToDevice);
        hipMemcpy(dmlist+nmass,grdata,grsize,hipMemcpyHostToDevice);
        myfree(grdata);
    }

    void save(std::vector<mass> &mlist,gpdata_t &mgp,ringdata_t &mrg){
        hipMemcpy(mlist.data(),dmlist,nmass*sizeof(mass),hipMemcpyDeviceToHost);
        for(auto &mgpi:mgp){
            mlist[mgpi.first].gpmodel=mgpi.second;
        }
        for(auto &mrgi:mrg){
            mlist[mrgi.first].ringmodel=mrgi.second;
        }
        mycudaFree((mass_state*)dmlist-26*nmass);
    }
};

__constant__ cuda_rungekutta_kernel_config dkf;

struct maccel_1{
    fast_mpmat C_potential;
    fast_mpvec naccel;
    fast_real phi;
};
struct maccel_2{
    fast_mpvec gaccel,daccel,dtorque;
    fast_real min_distance;
    fast_real max_influence;
};

//copied from geopotential.cpp, subroutines are added ``static __device__''
#if 1
//static const int_t Precompute_Table_size[1+Max_N]={0, 0, 10, 25, 46, 74, 110, 155, 210};
static __device__ int_t Precompute_Table_size(int_t n){
    return (n-1)*(36+n*(10+n))/6;
}

#if 0

template<const int_t n>
fast_mpvec sum_n(const fast_mpvec &r,const fast_mpvec *cn){
    int_t np=n+1,cp=0;
    fast_real xbase(1);
    fast_mpvec an(0);
    for(int_t px=0;px<=np;++px){
        fast_real ybase(xbase);
        for(int_t py=0;py<=np-px;++py){
            fast_real zbase(ybase);
            for(int_t pz=0;pz<np-px-py;++pz)zbase*=r.z;
            an+=zbase*cn[cp];
            ++cp;

            ybase*=r.y;
        }
        xbase*=r.x;
    }
    return an;
}

static const uint8_t j_table[]={1,2,1,2,4,0,4,1,2,4,1,2,1,2,1,4,0,4,0,1,2,1,4,0,1,1,2,1,2,1,2,4,0,4,0,4,1,2,1,2,4,0,4,1,2,4,1,2,1,2,1,2,1,4,0,4,0,4,0,1,2,1,2,1,4,0,4,0,1,2,1,4,0,1,1,2,1,2,1,2,1,2,4,0,4,0,4,0,4,1,2,1,2,1,2,4,0,4,0,4,1,2,1,2,4,0,4,1,2,4,1,2,1,2,1,2,1,2,1,4,0,4,0,4,0,4,0,1,2,1,2,1,2,1,4,0,4,0,4,0,1,2,1,2,1,4,0,4,0,1,2,1,4,0,1,1,2,1,2,1,2,1,2,1,2,4,0,4,0,4,0,4,0,4,1,2,1,2,1,2,1,2,4,0,4,0,4,0,4,1,2,1,2,1,2,4,0,4,0,4,1,2,1,2,4,0,4,1,2,4};
//summation over only terms include Jn coefficients
template<const int_t n>
fast_mpvec sum_Jn(const fast_mpvec &r,const fast_mpvec *cn){
    int_t np=n+1,cp=0;
    fast_real xbase(1);
    fast_mpvec an(0);
    const uint8_t *jt=j_table+Precompute_Table_size(n-1);
    for(int_t px=0;px<=np;++px){
        fast_real ybase(xbase);
        for(int_t py=0;py<=np-px;++py){
            fast_real zbase(ybase);
            for(int_t pz=0;pz<np-px-py;++pz)zbase*=r.z;
            if(jt[cp]&4)an.x+=zbase*cn[cp].x;
            if(jt[cp]&2)an.y+=zbase*cn[cp].y;
            if(jt[cp]&1)an.z+=zbase*cn[cp].z;
            ++cp;

            ybase*=r.y;
        }
        xbase*=r.x;
    }
    return an;
}

static fast_mpvec (*const sum_Jn_funlist[])(const fast_mpvec &,const fast_mpvec *)={
    nullptr,nullptr,
    sum_Jn<int_t(2)>,
    sum_Jn<int_t(3)>,
    sum_Jn<int_t(4)>,
    sum_Jn<int_t(5)>,
    sum_Jn<int_t(6)>,
    sum_Jn<int_t(7)>,
    sum_Jn<int_t(8)>
};

static fast_mpvec (*const sum_n_funlist[])(const fast_mpvec &,const fast_mpvec *)={
    nullptr,nullptr,
    sum_n<int_t(2)>,
    sum_n<int_t(3)>,
    sum_n<int_t(4)>,
    sum_n<int_t(5)>,
    sum_n<int_t(6)>,
    sum_n<int_t(7)>,
    sum_n<int_t(8)>
};

#else
static __device__ fast_mpvec sum_2(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);an+=base2.z*cn[0];an+=base0.y*base1.z*cn[1];an+=base1.y*base0.z*cn[2];an+=base2.y*cn[3];an+=base0.x*base1.z*cn[4];an+=base0.x*base0.y*base0.z*cn[5];an+=base0.x*base1.y*cn[6];an+=base1.x*base0.z*cn[7];an+=base1.x*base0.y*cn[8];an+=base2.x*cn[9];return an;
}

static __device__ fast_mpvec sum_3(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);an+=base3.z*cn[0];an+=base0.y*base2.z*cn[1];an+=base1.y*base1.z*cn[2];an+=base2.y*base0.z*cn[3];an+=base3.y*cn[4];an+=base0.x*base2.z*cn[5];an+=base0.x*base0.y*base1.z*cn[6];an+=base0.x*base1.y*base0.z*cn[7];an+=base0.x*base2.y*cn[8];an+=base1.x*base1.z*cn[9];an+=base1.x*base0.y*base0.z*cn[10];an+=base1.x*base1.y*cn[11];an+=base2.x*base0.z*cn[12];an+=base2.x*base0.y*cn[13];an+=base3.x*cn[14];return an;
}

static __device__ fast_mpvec sum_4(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);an+=base4.z*cn[0];an+=base0.y*base3.z*cn[1];an+=base1.y*base2.z*cn[2];an+=base2.y*base1.z*cn[3];an+=base3.y*base0.z*cn[4];an+=base4.y*cn[5];an+=base0.x*base3.z*cn[6];an+=base0.x*base0.y*base2.z*cn[7];an+=base0.x*base1.y*base1.z*cn[8];an+=base0.x*base2.y*base0.z*cn[9];an+=base0.x*base3.y*cn[10];an+=base1.x*base2.z*cn[11];an+=base1.x*base0.y*base1.z*cn[12];an+=base1.x*base1.y*base0.z*cn[13];an+=base1.x*base2.y*cn[14];an+=base2.x*base1.z*cn[15];an+=base2.x*base0.y*base0.z*cn[16];an+=base2.x*base1.y*cn[17];an+=base3.x*base0.z*cn[18];an+=base3.x*base0.y*cn[19];an+=base4.x*cn[20];return an;
}

static __device__ fast_mpvec sum_5(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);an+=base5.z*cn[0];an+=base0.y*base4.z*cn[1];an+=base1.y*base3.z*cn[2];an+=base2.y*base2.z*cn[3];an+=base3.y*base1.z*cn[4];an+=base4.y*base0.z*cn[5];an+=base5.y*cn[6];an+=base0.x*base4.z*cn[7];an+=base0.x*base0.y*base3.z*cn[8];an+=base0.x*base1.y*base2.z*cn[9];an+=base0.x*base2.y*base1.z*cn[10];an+=base0.x*base3.y*base0.z*cn[11];an+=base0.x*base4.y*cn[12];an+=base1.x*base3.z*cn[13];an+=base1.x*base0.y*base2.z*cn[14];an+=base1.x*base1.y*base1.z*cn[15];an+=base1.x*base2.y*base0.z*cn[16];an+=base1.x*base3.y*cn[17];an+=base2.x*base2.z*cn[18];an+=base2.x*base0.y*base1.z*cn[19];an+=base2.x*base1.y*base0.z*cn[20];an+=base2.x*base2.y*cn[21];an+=base3.x*base1.z*cn[22];an+=base3.x*base0.y*base0.z*cn[23];an+=base3.x*base1.y*cn[24];an+=base4.x*base0.z*cn[25];an+=base4.x*base0.y*cn[26];an+=base5.x*cn[27];return an;
}

static __device__ fast_mpvec sum_6(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);fast_mpvec base6(base5.x*r.x,base5.y*r.y,base5.z*r.z);an+=base6.z*cn[0];an+=base0.y*base5.z*cn[1];an+=base1.y*base4.z*cn[2];an+=base2.y*base3.z*cn[3];an+=base3.y*base2.z*cn[4];an+=base4.y*base1.z*cn[5];an+=base5.y*base0.z*cn[6];an+=base6.y*cn[7];an+=base0.x*base5.z*cn[8];an+=base0.x*base0.y*base4.z*cn[9];an+=base0.x*base1.y*base3.z*cn[10];an+=base0.x*base2.y*base2.z*cn[11];an+=base0.x*base3.y*base1.z*cn[12];an+=base0.x*base4.y*base0.z*cn[13];an+=base0.x*base5.y*cn[14];an+=base1.x*base4.z*cn[15];an+=base1.x*base0.y*base3.z*cn[16];an+=base1.x*base1.y*base2.z*cn[17];an+=base1.x*base2.y*base1.z*cn[18];an+=base1.x*base3.y*base0.z*cn[19];an+=base1.x*base4.y*cn[20];an+=base2.x*base3.z*cn[21];an+=base2.x*base0.y*base2.z*cn[22];an+=base2.x*base1.y*base1.z*cn[23];an+=base2.x*base2.y*base0.z*cn[24];an+=base2.x*base3.y*cn[25];an+=base3.x*base2.z*cn[26];an+=base3.x*base0.y*base1.z*cn[27];an+=base3.x*base1.y*base0.z*cn[28];an+=base3.x*base2.y*cn[29];an+=base4.x*base1.z*cn[30];an+=base4.x*base0.y*base0.z*cn[31];an+=base4.x*base1.y*cn[32];an+=base5.x*base0.z*cn[33];an+=base5.x*base0.y*cn[34];an+=base6.x*cn[35];return an;
}

static __device__ fast_mpvec sum_7(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);fast_mpvec base6(base5.x*r.x,base5.y*r.y,base5.z*r.z);fast_mpvec base7(base6.x*r.x,base6.y*r.y,base6.z*r.z);an+=base7.z*cn[0];an+=base0.y*base6.z*cn[1];an+=base1.y*base5.z*cn[2];an+=base2.y*base4.z*cn[3];an+=base3.y*base3.z*cn[4];an+=base4.y*base2.z*cn[5];an+=base5.y*base1.z*cn[6];an+=base6.y*base0.z*cn[7];an+=base7.y*cn[8];an+=base0.x*base6.z*cn[9];an+=base0.x*base0.y*base5.z*cn[10];an+=base0.x*base1.y*base4.z*cn[11];an+=base0.x*base2.y*base3.z*cn[12];an+=base0.x*base3.y*base2.z*cn[13];an+=base0.x*base4.y*base1.z*cn[14];an+=base0.x*base5.y*base0.z*cn[15];an+=base0.x*base6.y*cn[16];an+=base1.x*base5.z*cn[17];an+=base1.x*base0.y*base4.z*cn[18];an+=base1.x*base1.y*base3.z*cn[19];an+=base1.x*base2.y*base2.z*cn[20];an+=base1.x*base3.y*base1.z*cn[21];an+=base1.x*base4.y*base0.z*cn[22];an+=base1.x*base5.y*cn[23];an+=base2.x*base4.z*cn[24];an+=base2.x*base0.y*base3.z*cn[25];an+=base2.x*base1.y*base2.z*cn[26];an+=base2.x*base2.y*base1.z*cn[27];an+=base2.x*base3.y*base0.z*cn[28];an+=base2.x*base4.y*cn[29];an+=base3.x*base3.z*cn[30];an+=base3.x*base0.y*base2.z*cn[31];an+=base3.x*base1.y*base1.z*cn[32];an+=base3.x*base2.y*base0.z*cn[33];an+=base3.x*base3.y*cn[34];an+=base4.x*base2.z*cn[35];an+=base4.x*base0.y*base1.z*cn[36];an+=base4.x*base1.y*base0.z*cn[37];an+=base4.x*base2.y*cn[38];an+=base5.x*base1.z*cn[39];an+=base5.x*base0.y*base0.z*cn[40];an+=base5.x*base1.y*cn[41];an+=base6.x*base0.z*cn[42];an+=base6.x*base0.y*cn[43];an+=base7.x*cn[44];return an;
}

static __device__ fast_mpvec sum_8(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);fast_mpvec base6(base5.x*r.x,base5.y*r.y,base5.z*r.z);fast_mpvec base7(base6.x*r.x,base6.y*r.y,base6.z*r.z);fast_mpvec base8(base7.x*r.x,base7.y*r.y,base7.z*r.z);an+=base8.z*cn[0];an+=base0.y*base7.z*cn[1];an+=base1.y*base6.z*cn[2];an+=base2.y*base5.z*cn[3];an+=base3.y*base4.z*cn[4];an+=base4.y*base3.z*cn[5];an+=base5.y*base2.z*cn[6];an+=base6.y*base1.z*cn[7];an+=base7.y*base0.z*cn[8];an+=base8.y*cn[9];an+=base0.x*base7.z*cn[10];an+=base0.x*base0.y*base6.z*cn[11];an+=base0.x*base1.y*base5.z*cn[12];an+=base0.x*base2.y*base4.z*cn[13];an+=base0.x*base3.y*base3.z*cn[14];an+=base0.x*base4.y*base2.z*cn[15];an+=base0.x*base5.y*base1.z*cn[16];an+=base0.x*base6.y*base0.z*cn[17];an+=base0.x*base7.y*cn[18];an+=base1.x*base6.z*cn[19];an+=base1.x*base0.y*base5.z*cn[20];an+=base1.x*base1.y*base4.z*cn[21];an+=base1.x*base2.y*base3.z*cn[22];an+=base1.x*base3.y*base2.z*cn[23];an+=base1.x*base4.y*base1.z*cn[24];an+=base1.x*base5.y*base0.z*cn[25];an+=base1.x*base6.y*cn[26];an+=base2.x*base5.z*cn[27];an+=base2.x*base0.y*base4.z*cn[28];an+=base2.x*base1.y*base3.z*cn[29];an+=base2.x*base2.y*base2.z*cn[30];an+=base2.x*base3.y*base1.z*cn[31];an+=base2.x*base4.y*base0.z*cn[32];an+=base2.x*base5.y*cn[33];an+=base3.x*base4.z*cn[34];an+=base3.x*base0.y*base3.z*cn[35];an+=base3.x*base1.y*base2.z*cn[36];an+=base3.x*base2.y*base1.z*cn[37];an+=base3.x*base3.y*base0.z*cn[38];an+=base3.x*base4.y*cn[39];an+=base4.x*base3.z*cn[40];an+=base4.x*base0.y*base2.z*cn[41];an+=base4.x*base1.y*base1.z*cn[42];an+=base4.x*base2.y*base0.z*cn[43];an+=base4.x*base3.y*cn[44];an+=base5.x*base2.z*cn[45];an+=base5.x*base0.y*base1.z*cn[46];an+=base5.x*base1.y*base0.z*cn[47];an+=base5.x*base2.y*cn[48];an+=base6.x*base1.z*cn[49];an+=base6.x*base0.y*base0.z*cn[50];an+=base6.x*base1.y*cn[51];an+=base7.x*base0.z*cn[52];an+=base7.x*base0.y*cn[53];an+=base8.x*cn[54];return an;
}

static __device__ fast_mpvec sum_J2(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_real zbase;an.z+=base2.z*cn[0].z;an.y+=base0.y*base1.z*cn[1].y;an.z+=base1.y*base0.z*cn[2].z;an.y+=base2.y*cn[3].y;an.x+=base0.x*base1.z*cn[4].x;an.x+=base0.x*base1.y*cn[6].x;an.z+=base1.x*base0.z*cn[7].z;an.y+=base1.x*base0.y*cn[8].y;an.x+=base2.x*cn[9].x;return an;
}

static __device__ fast_mpvec sum_J3(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_real zbase;an.z+=base3.z*cn[0].z;an.y+=base0.y*base2.z*cn[1].y;an.z+=base1.y*base1.z*cn[2].z;an.y+=base2.y*base0.z*cn[3].y;an.z+=base3.y*cn[4].z;an.x+=base0.x*base2.z*cn[5].x;an.x+=base0.x*base1.y*base0.z*cn[7].x;an.z+=base1.x*base1.z*cn[9].z;an.y+=base1.x*base0.y*base0.z*cn[10].y;an.z+=base1.x*base1.y*cn[11].z;an.x+=base2.x*base0.z*cn[12].x;an.z+=base3.x*cn[14].z;return an;
}

static __device__ fast_mpvec sum_J4(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_real zbase;an.z+=base4.z*cn[0].z;an.y+=base0.y*base3.z*cn[1].y;an.z+=base1.y*base2.z*cn[2].z;an.y+=base2.y*base1.z*cn[3].y;an.z+=base3.y*base0.z*cn[4].z;an.y+=base4.y*cn[5].y;an.x+=base0.x*base3.z*cn[6].x;an.x+=base0.x*base1.y*base1.z*cn[8].x;an.x+=base0.x*base3.y*cn[10].x;an.z+=base1.x*base2.z*cn[11].z;an.y+=base1.x*base0.y*base1.z*cn[12].y;an.z+=base1.x*base1.y*base0.z*cn[13].z;an.y+=base1.x*base2.y*cn[14].y;an.x+=base2.x*base1.z*cn[15].x;an.x+=base2.x*base1.y*cn[17].x;an.z+=base3.x*base0.z*cn[18].z;an.y+=base3.x*base0.y*cn[19].y;an.x+=base4.x*cn[20].x;return an;
}

static __device__ fast_mpvec sum_J5(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);fast_real zbase;an.z+=base5.z*cn[0].z;an.y+=base0.y*base4.z*cn[1].y;an.z+=base1.y*base3.z*cn[2].z;an.y+=base2.y*base2.z*cn[3].y;an.z+=base3.y*base1.z*cn[4].z;an.y+=base4.y*base0.z*cn[5].y;an.z+=base5.y*cn[6].z;an.x+=base0.x*base4.z*cn[7].x;an.x+=base0.x*base1.y*base2.z*cn[9].x;an.x+=base0.x*base3.y*base0.z*cn[11].x;an.z+=base1.x*base3.z*cn[13].z;an.y+=base1.x*base0.y*base2.z*cn[14].y;an.z+=base1.x*base1.y*base1.z*cn[15].z;an.y+=base1.x*base2.y*base0.z*cn[16].y;an.z+=base1.x*base3.y*cn[17].z;an.x+=base2.x*base2.z*cn[18].x;an.x+=base2.x*base1.y*base0.z*cn[20].x;an.z+=base3.x*base1.z*cn[22].z;an.y+=base3.x*base0.y*base0.z*cn[23].y;an.z+=base3.x*base1.y*cn[24].z;an.x+=base4.x*base0.z*cn[25].x;an.z+=base5.x*cn[27].z;return an;
}

static __device__ fast_mpvec sum_J6(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);fast_mpvec base6(base5.x*r.x,base5.y*r.y,base5.z*r.z);fast_real zbase;an.z+=base6.z*cn[0].z;an.y+=base0.y*base5.z*cn[1].y;an.z+=base1.y*base4.z*cn[2].z;an.y+=base2.y*base3.z*cn[3].y;an.z+=base3.y*base2.z*cn[4].z;an.y+=base4.y*base1.z*cn[5].y;an.z+=base5.y*base0.z*cn[6].z;an.y+=base6.y*cn[7].y;an.x+=base0.x*base5.z*cn[8].x;an.x+=base0.x*base1.y*base3.z*cn[10].x;an.x+=base0.x*base3.y*base1.z*cn[12].x;an.x+=base0.x*base5.y*cn[14].x;an.z+=base1.x*base4.z*cn[15].z;an.y+=base1.x*base0.y*base3.z*cn[16].y;an.z+=base1.x*base1.y*base2.z*cn[17].z;an.y+=base1.x*base2.y*base1.z*cn[18].y;an.z+=base1.x*base3.y*base0.z*cn[19].z;an.y+=base1.x*base4.y*cn[20].y;an.x+=base2.x*base3.z*cn[21].x;an.x+=base2.x*base1.y*base1.z*cn[23].x;an.x+=base2.x*base3.y*cn[25].x;an.z+=base3.x*base2.z*cn[26].z;an.y+=base3.x*base0.y*base1.z*cn[27].y;an.z+=base3.x*base1.y*base0.z*cn[28].z;an.y+=base3.x*base2.y*cn[29].y;an.x+=base4.x*base1.z*cn[30].x;an.x+=base4.x*base1.y*cn[32].x;an.z+=base5.x*base0.z*cn[33].z;an.y+=base5.x*base0.y*cn[34].y;an.x+=base6.x*cn[35].x;return an;
}

static __device__ fast_mpvec sum_J7(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);fast_mpvec base6(base5.x*r.x,base5.y*r.y,base5.z*r.z);fast_mpvec base7(base6.x*r.x,base6.y*r.y,base6.z*r.z);fast_real zbase;an.z+=base7.z*cn[0].z;an.y+=base0.y*base6.z*cn[1].y;an.z+=base1.y*base5.z*cn[2].z;an.y+=base2.y*base4.z*cn[3].y;an.z+=base3.y*base3.z*cn[4].z;an.y+=base4.y*base2.z*cn[5].y;an.z+=base5.y*base1.z*cn[6].z;an.y+=base6.y*base0.z*cn[7].y;an.z+=base7.y*cn[8].z;an.x+=base0.x*base6.z*cn[9].x;an.x+=base0.x*base1.y*base4.z*cn[11].x;an.x+=base0.x*base3.y*base2.z*cn[13].x;an.x+=base0.x*base5.y*base0.z*cn[15].x;an.z+=base1.x*base5.z*cn[17].z;an.y+=base1.x*base0.y*base4.z*cn[18].y;an.z+=base1.x*base1.y*base3.z*cn[19].z;an.y+=base1.x*base2.y*base2.z*cn[20].y;an.z+=base1.x*base3.y*base1.z*cn[21].z;an.y+=base1.x*base4.y*base0.z*cn[22].y;an.z+=base1.x*base5.y*cn[23].z;an.x+=base2.x*base4.z*cn[24].x;an.x+=base2.x*base1.y*base2.z*cn[26].x;an.x+=base2.x*base3.y*base0.z*cn[28].x;an.z+=base3.x*base3.z*cn[30].z;an.y+=base3.x*base0.y*base2.z*cn[31].y;an.z+=base3.x*base1.y*base1.z*cn[32].z;an.y+=base3.x*base2.y*base0.z*cn[33].y;an.z+=base3.x*base3.y*cn[34].z;an.x+=base4.x*base2.z*cn[35].x;an.x+=base4.x*base1.y*base0.z*cn[37].x;an.z+=base5.x*base1.z*cn[39].z;an.y+=base5.x*base0.y*base0.z*cn[40].y;an.z+=base5.x*base1.y*cn[41].z;an.x+=base6.x*base0.z*cn[42].x;an.z+=base7.x*cn[44].z;return an;
}

static __device__ fast_mpvec sum_J8(const fast_mpvec &r,const fast_mpvec *cn){
    fast_mpvec base0(r),an(0);fast_mpvec base1(base0.x*r.x,base0.y*r.y,base0.z*r.z);fast_mpvec base2(base1.x*r.x,base1.y*r.y,base1.z*r.z);fast_mpvec base3(base2.x*r.x,base2.y*r.y,base2.z*r.z);fast_mpvec base4(base3.x*r.x,base3.y*r.y,base3.z*r.z);fast_mpvec base5(base4.x*r.x,base4.y*r.y,base4.z*r.z);fast_mpvec base6(base5.x*r.x,base5.y*r.y,base5.z*r.z);fast_mpvec base7(base6.x*r.x,base6.y*r.y,base6.z*r.z);fast_mpvec base8(base7.x*r.x,base7.y*r.y,base7.z*r.z);fast_real zbase;an.z+=base8.z*cn[0].z;an.y+=base0.y*base7.z*cn[1].y;an.z+=base1.y*base6.z*cn[2].z;an.y+=base2.y*base5.z*cn[3].y;an.z+=base3.y*base4.z*cn[4].z;an.y+=base4.y*base3.z*cn[5].y;an.z+=base5.y*base2.z*cn[6].z;an.y+=base6.y*base1.z*cn[7].y;an.z+=base7.y*base0.z*cn[8].z;an.y+=base8.y*cn[9].y;an.x+=base0.x*base7.z*cn[10].x;an.x+=base0.x*base1.y*base5.z*cn[12].x;an.x+=base0.x*base3.y*base3.z*cn[14].x;an.x+=base0.x*base5.y*base1.z*cn[16].x;an.x+=base0.x*base7.y*cn[18].x;an.z+=base1.x*base6.z*cn[19].z;an.y+=base1.x*base0.y*base5.z*cn[20].y;an.z+=base1.x*base1.y*base4.z*cn[21].z;an.y+=base1.x*base2.y*base3.z*cn[22].y;an.z+=base1.x*base3.y*base2.z*cn[23].z;an.y+=base1.x*base4.y*base1.z*cn[24].y;an.z+=base1.x*base5.y*base0.z*cn[25].z;an.y+=base1.x*base6.y*cn[26].y;an.x+=base2.x*base5.z*cn[27].x;an.x+=base2.x*base1.y*base3.z*cn[29].x;an.x+=base2.x*base3.y*base1.z*cn[31].x;an.x+=base2.x*base5.y*cn[33].x;an.z+=base3.x*base4.z*cn[34].z;an.y+=base3.x*base0.y*base3.z*cn[35].y;an.z+=base3.x*base1.y*base2.z*cn[36].z;an.y+=base3.x*base2.y*base1.z*cn[37].y;an.z+=base3.x*base3.y*base0.z*cn[38].z;an.y+=base3.x*base4.y*cn[39].y;an.x+=base4.x*base3.z*cn[40].x;an.x+=base4.x*base1.y*base1.z*cn[42].x;an.x+=base4.x*base3.y*cn[44].x;an.z+=base5.x*base2.z*cn[45].z;an.y+=base5.x*base0.y*base1.z*cn[46].y;an.z+=base5.x*base1.y*base0.z*cn[47].z;an.y+=base5.x*base2.y*cn[48].y;an.x+=base6.x*base1.z*cn[49].x;an.x+=base6.x*base1.y*cn[51].x;an.z+=base7.x*base0.z*cn[52].z;an.y+=base7.x*base0.y*cn[53].y;an.x+=base8.x*cn[54].x;return an;
}


static __device__ fast_mpvec (*const sum_Jn_funlist[])(const fast_mpvec &,const fast_mpvec *)={
    nullptr,nullptr,
    sum_J2,
    sum_J3,
    sum_J4,
    sum_J5,
    sum_J6,
    sum_J7,
    sum_J8
};

static __device__ fast_mpvec (*const sum_n_funlist[])(const fast_mpvec &,const fast_mpvec *)={
    nullptr,nullptr,
    sum_2,
    sum_3,
    sum_4,
    sum_5,
    sum_6,
    sum_7,
    sum_8
};
#endif


__device__ fast_mpvec geopotential::cuda_sum(fast_real R,fast_mpvec r,int_t N_start,int_t N_end) const{
    fast_real x=r.x,y=r.y,z=r.z;
    fast_real rr2=1/(r%r);
    fast_real rr=sqrt(rr2);
    fast_real R_r2=R*rr2;

    if(N_start<2)N_start=2;
    if(N_end<0||N_end>N)N_end=N;

    fast_mpvec a(0);
    for(int_t n=N_end;n>=1;--n){
        if(n>=N_start){
            a+=(n>Nt?sum_Jn_funlist:sum_n_funlist)[n](r,c_table+Precompute_Table_size(n-1));
        }
        a*=R_r2;
    }
    a*=-rr2*rr;

    return a;
}
#endif

//copied from ring.cpp, subroutines are added ``static __device__''
#if 1

#define CONST_TABLE static __device__ const
#include"disk_approx.impl"
static __device__ float fzcorr(float x,float y,int kh){
    const size_t N=32;

    x*=N;
    y*=N;
    int i=floor(x),j=floor(y);
    j-=kh*N;
    if(i<0)i=0;
    if(j<0)j=0;
    if(i>=N)i=N-1;
    if(j>=N)j=N-1;
    j+=kh*N;
    x-=i;
    y-=j;
    const float *p=(const float *)disk_approx_table+16*(2*N*i+j);
    return  p[0]+x*(p[4]+x*(p[8]+x*p[12]))
        +y*(p[1]+x*(p[5]+x*(p[9]+x*p[13]))
            +y*(p[2]+x*(p[6]+x*(p[10]+x*p[14]))
                +y*(p[3]+x*(p[7]+x*(p[11]+x*p[15])))));
}
static __device__ double padesum(double x,const double coef[][2],int n){
    double resn=0,resd=0;
    while(n>0){
        --n;
        resn=x*resn+coef[n][0];
        resd=x*resd+coef[n][1];
    }
    return resn/resd;
}
#define PadeSum(x,coef) padesum(x,coef,sizeof(coef)/sizeof(coef[0]))

__device__ fast_mpvec ring::cuda_sum(fast_mpvec r) const{
    const double pi=3.1415926535897932;
    const double xserkk[][2]={{-4228.1846570321674587,6387.9414439779279738},{-13276.372802884782407,16984.953742612298093},{-11316.549465243054011,8997.7319379726254196},{-3242.2867356286923985,403.15182320396046619},{-704.60633921130372399,-5.7789477668119521439}};
    const double xserkm[][2]={{-2530.8255093002657219,7950.8733392382639504},{-5471.7803365810451727,17189.101860053144376},{-1624.0389725420726306,7100.5562444641275465},{1164.6058458813108946,532.54915854738706820},{270.03897254207263065,-5.0806023029229406554}};
    const double xser0[][2]={{3216.9908772759482762,32768.000000000000000},{-6727.1375380948220662,-93098.060290997395032},{4534.8257724052420971,96814.901233532309690},{-1004.5251977686921953,-43973.540867249017404},{17.915064837735955511,7754.0323173314244943},{-0.039157108395559676838,-224.00490789882410101}};

    const double kserkk[][2]={{-2373.3522804139070577,1124.9866194216978637},{-8300.7524798211745830,17215.217000406477760},{15279.068333441193395,26590.060415743810476},{-4354.4701057632348105,-9776.5071563707548874},{-250.49346744287694398,-2385.7568792012312128}};
    const double kserkm[][2]={{-523.02599873176918199,290.10526315789473684},{-5485.2200380469245403,5336.9790741915028535},{-8338.1052631578947368,15206.361445783132530},{-2017.9378566899175650,10515.743817374762207},{-19.710843373493975904,1418.8103994927076728}};
    const double kser0[][2]={{51471.854036415172419,32768.000000000000000},{-119621.18573838632696,-84345.212035111671234},{96754.491735189724987,78074.125512775430927},{-31810.735585315211544,-31108.829182924968396},{3678.1383813624327949,4926.4461273804873303},{-74.040670509817904983,-206.66608791871105442}};

    bool psign;
    if(psign=r.z<0)r.z=-r.z;

    const double re2(r.x*r.x+r.y*r.y);
    const double re=sqrt(re2);
    const double rz2(r.z*r.z);
    const double r2=re2+rz2;
    const double rz(r.z);

    fast_mpvec ret(0);

    for(int_t i=0;i<N;++i){
        double R=c_table[i].R;
        double H=c_table[i].H;

        double reR=re*R;
        double R2=R*R,H2,fzh=1;
        double k2dre=4/(r2+R2+2*reR);
        if(H>0){
            H2=H*H;
            k2dre-=H2*(k2dre*k2dre)*(1-k2dre*rz2)/48;
            fzh=(re>R?re-R:0);
            fzh=4*(rz2+fzh*fzh);
            fzh=sqrt(fzh/(fzh+H2));
        }
        double k2=reR*k2dre;
        k2dre*=R2*sqrt(k2dre);
        double phi=atan2(rz,re-R);
        double k,lk,fx;
        bool lkh=k2>=0.45;
        if(lkh){
            k=sqrt(k2);lk=log((1-k2)/(8*(1+k)));
            fx=(PadeSum(k,xserkk)+lk*PadeSum(k,xserkm))/(k2*k2);
        }
        else{
            fx=PadeSum(k2,xser0);
        }
        fx*=k2dre;

        //approx for fz=diskfz_approx(phi,k2)
        double fz;
        double s=sin(phi),c=cos(phi);
        bool kh=k2>=.5;
        bool exphi;

        if(kh){
            const double phimax=(pi/2);
            exphi=c>0;
            double ck=1-k2,lk=log(ck/16),sk=sqrt(ck);
            double cc=exphi?-c:c;
            double cphi=exphi?pi-phi:phi;
            fz=cphi-(s/8)*((lk*(k2-5)-2*ck)*sk+2*cc*(1+lk)*ck);
            fz*=fzcorr((exphi?phi:pi-phi)*(1/phimax),k2*2,kh);
        }
        else{
            const double phimax=(809./512);
            exphi=phi>phimax;
            double cr2=exphi?R2+rz2:r2;
            double cRdr2=(exphi?re2:R2)/cr2;
            fz=(pi/8)*(4-3*cRdr2)*(rz/sqrt(cr2)*cRdr2);
            fz*=fzcorr((exphi?pi-phi:phi)*(1/phimax),k2*2,kh);
        }

        if(exphi){
            double ek;
            if(lkh){
                ek=PadeSum(k,kserkk)+lk*PadeSum(k,kserkm);
            }
            else{
                ek=PadeSum(k2,kser0);
            }
            fz=pi-2*sqrt(1-k2)*ek*s-fz;
        }

        double Gs=c_table[i].Gs;
        fx*=Gs;
        fz*=Gs;

        ret+=fast_mpvec(-4*fx*r.x,-4*fx*r.y,-2*fz*fzh);
    }

    if(psign)ret.z=-ret.z;

    return ret;
}
#endif

#include"RungeKutta.impl"


extern __shared__ char sharedMem[];
void __device__ accel_0(){//deform

    const fast_real c=299792458;
    const fast_real c2=c*c;

    int i0=blockIdx.x*dkf.mass_per_block;
    for(int di=0;di<dkf.mass_per_block;++di){
        int i=di+i0;
        if(i<dkf.nmass){
            mass &mi=dkf.dmlist[i];

            int_t max_iter=4;
            do{
                maccel_1 *tpmi=(maccel_1 *)sharedMem+threadIdx.x;
                tpmi[0].phi=0;
                tpmi[0].naccel=0;
                tpmi[0].C_potential=0;
                for(int dj=0;dj<dkf.nmass;dj+=blockDim.x){
                    int j=dj+threadIdx.x;
                    if(j<dkf.nmass&&i!=j){
                        mass &mj=dkf.dmlist[j];
                        //---accel-prepare
                        fast_mpvec r=mj.r-mi.r;
                        fast_mpvec v=mj.v-mi.v;
                        fast_real rr2=1/(r%r);
                        fast_real rr=sqrt(rr2);
                        fast_real tp_dphi=rr*mj.GM;
                        fast_real tp_dg=rr2*tp_dphi;

                        tpmi[0].phi-=tp_dphi;
                        tpmi[0].naccel+=tp_dg*r;

                        //damped tidal deformation matrix
                        fast_real fmj=-3*tp_dg;
                        fast_mpvec dw=r*(mi.w*r-v)*rr2;
                        fast_real dw2=dw%dw,dw1=sqrt(dw2),dwt=dw1*mi.tide_delay*mj.tide_delay_factor;
                        fast_real
                            ecwt2=1+4*dwt*dwt,
                            secwt2=sqrt(ecwt2),
                            x0=sqrt((1+secwt2)/(2*ecwt2)),
                            y0=dwt*sqrt(2/(ecwt2*(1+secwt2))),
                            z02=dwt*dwt*(2/(ecwt2+secwt2));
                        fast_mpvec
                            dr=r*x0+dw*r*(y0/dw1);
                        tpmi[0].C_potential+=(
                            fast_mpmat(fast_real(1)/3-z02)
                            -fast_mpmat(dr*rr2,dr)
                            +fast_mpmat(dw*(z02/dw2),dw)
                            )*(fmj*mi.k2);
                    }
                }

                for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                    __syncthreads();
                    if(threadIdx.x<wing){
                        tpmi[0].phi+=tpmi[wing].phi;
                        tpmi[0].naccel+=tpmi[wing].naccel;
                        tpmi[0].C_potential+=tpmi[wing].C_potential;
                    }
                }
                __syncthreads();
                if(threadIdx.x<1){
                    mi.phi=tpmi[0].phi+tpmi[1].phi;
                    mi.naccel=tpmi[0].naccel+tpmi[1].naccel;
                    mi.C_potential=tpmi[0].C_potential+tpmi[1].C_potential;
                }
                __syncthreads();

                __shared__ bool should_break;
                //angular accelerate
                if(threadIdx.x==0){
                    fast_real w2=mi.w%mi.w;
                    mi.C_potential+=fast_mpmat(w2*mi.k2r/3)-fast_mpmat(mi.w*mi.k2r,mi.w);
                    mi.C_potential*=mi.R*mi.R2/(2*mi.GM);

                    fast_mpmat fmis(mi.s);
                    fast_mpmat mc(mi.C_static);
                    mc.x.x+=mi.exJ2/2;
                    mc.y.y+=mi.exJ2/2;
                    mc.z.z-=mi.exJ2;
                    mc=fmis.toworld(mc);
                    mi.C_potential+=mc;
                    mi.GI=2*mi.R2/3*(fast_mpmat(mi.A)-mi.C_potential);
                    fast_mpvec oldw=mi.w;
                    mi.w=mi.GI.inverse()%(fast_mpvec(mi.GL));
                    should_break=((mi.w-oldw).norm()<1e-9*oldw.norm());
                }
                __syncthreads();
                if(should_break)break;
            } while(--max_iter);
            if(threadIdx.x==0){
                mi.beta=fast_mpvec(mi.v)/c;
                mi.beta2=mi.beta%mi.beta;
                mi.phi/=c2;
                mi.naccel/=c2;
            }
        }
    }
}

void __device__ accel_1(){//accel
    const fast_real c=299792458;
    const fast_real c2=c*c;

    int i0=blockIdx.x*dkf.mass_per_block;
    for(int di=0;di<dkf.mass_per_block;++di){
        int i=di+i0;
        if(i<dkf.nmass){
            mass &mi=dkf.dmlist[i];
            maccel_2 *tpmi=(maccel_2*)sharedMem+threadIdx.x;
            tpmi[0].gaccel=0;
            tpmi[0].daccel=0;
            tpmi[0].dtorque=0;
            tpmi[0].min_distance=0;
            tpmi[0].max_influence=0;
            for(int dj=0;dj<dkf.nmass;dj+=blockDim.x){
                int j=dj+threadIdx.x;
                if(j<dkf.nmass&&i!=j){
                    mass &mj=dkf.dmlist[j];
                    //---accel
                    fast_mpvec r=mj.r-mi.r;
                    fast_real rr2=1/(r%r);
                    fast_real rr=sqrt(rr2);
                    fast_real rr3=rr*rr2;

                    //start post-newtonian correction
                    fast_real tp_dphi=rr*mj.GM;
                    fast_real tp_dg=rr2*tp_dphi;
                    fast_real rbj=r%mj.beta;
                    fast_real tp_rbjrr2=rbj*rr;
                    tp_rbjrr2*=tp_rbjrr2;
                    fast_real delta1=4*mi.phi+mj.phi+mi.beta2+2*mj.beta2-4*(mi.beta%mj.beta)+(r%mj.naccel-3*tp_rbjrr2)/2;
                    fast_mpvec b=mj.beta-mi.beta;
                    tpmi[0].gaccel+=7*tp_dphi/2*mj.naccel+tp_dg*delta1*r+tp_dg*(rbj-r%b*4)*b;
                    //end post-newtonian correction
                    tpmi[0].min_distance=cuda_max(tpmi[0].min_distance,rr);
                    tpmi[0].max_influence=cuda_max(tpmi[0].max_influence,tp_dg);
                    //rotational & tidal deformation: gravity, torque
                    fast_mpvec Cr=mi.C_potential%r;
                    fast_mpvec dg=mj.GM*rr3*rr2*mi.R2*(r%Cr*5*rr2*r-(Cr+Cr));
                    tpmi[0].daccel+=dg;
                    tpmi[0].dtorque+=r*dg;
                    //to avoid reduce cross thread block, re-calculate daccel instead of using anti-force
                    fast_mpvec Cjr=mj.C_potential%r;
                    tpmi[0].daccel+=mj.GM*rr3*rr2*mj.R2*(r%Cjr*5*rr2*r-(Cjr+Cjr));
                    //start lense thirring
                    fast_real rcr32=2*mj.GM/c*rr3;
                    fast_mpvec GL=mi.GL;
                    fast_mpvec GLb=GL*b;
                    tpmi[0].daccel-=rcr32*(GLb-3*(GLb%r)*rr2*r);
                    tpmi[0].dtorque-=rcr32*(GL*(r*b));
                    fast_mpvec GLj=mj.GL;
                    tpmi[0].daccel+=(rcr32*(GLj-3*(GLj%r)*rr2*r))*b;
                    //end lense thirring

                    //start radiation pressure
                    tpmi[0].daccel+=mj.lum*mi.rR2_4Mc*rr3*r;
                    //end radiation pressure

                    //---accel
                    //higher harmonics will be done later
                }
            }

            for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                __syncthreads();
                if(threadIdx.x<wing){
                    tpmi[0].gaccel+=tpmi[wing].gaccel;
                    tpmi[0].daccel+=tpmi[wing].daccel;
                    tpmi[0].dtorque+=tpmi[wing].dtorque;
                    tpmi[0].min_distance=cuda_max(tpmi[0].min_distance,tpmi[wing].min_distance);
                    tpmi[0].max_influence=cuda_max(tpmi[0].max_influence,tpmi[wing].max_influence);
                }
            }
            __syncthreads();
            if(threadIdx.x<1){
                mi.gaccel=tpmi[0].gaccel+tpmi[1].gaccel;
                mi.daccel=tpmi[0].daccel+tpmi[1].daccel;
                mi.dtorque=tpmi[0].dtorque+tpmi[1].dtorque;
                tpmi[0].min_distance=cuda_max(tpmi[0].min_distance,tpmi[1].min_distance);
                mi.min_distance=cuda_max(mi.min_distance,tpmi[0].min_distance);
                tpmi[0].max_influence=cuda_max(tpmi[0].max_influence,tpmi[1].max_influence);
                mi.max_influence=cuda_max(mi.max_influence,tpmi[0].max_influence);
            }
            __syncthreads();
        }
    }
}

void __device__ accel_2(){//higher harmonics
    int mn=dkf.nmass;
    int i0=blockIdx.x*dkf.mass_per_block;

    for(int j=0;j<dkf.nmass;++j){
        mass &mi=dkf.dmlist[j];
        if(mi.gpmodel||mi.ringmodel){
            maccel_2 *tpmi=(maccel_2*)sharedMem+threadIdx.x;
            tpmi[0].daccel=0;
            tpmi[0].dtorque=0;

            for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
                int i=i0+di+threadIdx.x;
                if(i<mn&&di+threadIdx.x<dkf.mass_per_block&&i!=j){
                    mass &mj=dkf.dmlist[i];
                    fast_mpvec an(0);
                    fast_mpvec r=mj.r-mi.r;
                    if(mi.gpmodel){
                        fast_mpmat fmis(mi.s);
                        fast_mpvec lr=fmis.tolocal(r);
                        an+=fmis.toworld(mi.gpmodel->cuda_sum(mi.R,lr));
                    }
                    if(mi.ringmodel){
                        fast_mpvec migl=mi.GL;
                        fast_mpmat fgls(migl.perpunit(),0,migl/migl.norm());
                        fast_mpvec lr=fgls.tolocal(r);
                        an+=fgls.toworld(mi.ringmodel->cuda_sum(lr));
                    }

                    mj.daccel+=mi.GM*an;
                    tpmi[0].daccel-=mj.GM*an;
                    tpmi[0].dtorque-=mj.GM*(r*an);


                }
            }
            //the initial wing should be lift to 2's power and the mask should be adjusted.
            for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                __syncthreads();
                if(threadIdx.x<wing){
                    tpmi[0].daccel+=tpmi[wing].daccel;
                    tpmi[0].dtorque+=tpmi[wing].dtorque;
                }
            }
            __syncthreads();
            if(threadIdx.x<1){
                //Note: by construction, nblocks <= nmass
                //      so we store partial acceleration per block in global mlist memory
                mass &mbi=dkf.dmlist[blockIdx.x];
                mbi.idaccel=tpmi[0].daccel+tpmi[1].daccel;
                mbi.idtorque=tpmi[0].dtorque+tpmi[1].dtorque;
            }
            __syncthreads();


            cooperative_groups::grid_group grid=cooperative_groups::this_grid();
            grid.sync();
            if(blockIdx.x==0){
                // block 0
                tpmi[0].daccel=0;
                tpmi[0].dtorque=0;
                for(int i=threadIdx.x;i<gridDim.x;i+=blockDim.x){
                    mass &mj=dkf.dmlist[i];
                    tpmi[0].daccel+=mj.idaccel;
                    tpmi[0].dtorque+=mj.idtorque;
                }

                for(int wing=blockDim.x>>1;wing>1;wing>>=1){
                    __syncthreads();
                    if(threadIdx.x<wing){
                        tpmi[0].daccel+=tpmi[wing].daccel;
                        tpmi[0].dtorque+=tpmi[wing].dtorque;
                    }
                }
                __syncthreads();
                if(threadIdx.x<1){
                    mi.daccel+=tpmi[0].daccel+tpmi[1].daccel;
                    mi.dtorque+=tpmi[0].dtorque+tpmi[1].dtorque;
                }
                __syncthreads();

            }
            grid.sync();
        }
    }
}

void __device__ Cuda_accel(){
    const fast_real c=299792458;
    const fast_real c2=c*c;
    cooperative_groups::grid_group grid=cooperative_groups::this_grid();

    mass *x=dkf.dmlist;
    int mn=dkf.nmass;
    int i0=blockIdx.x*dkf.mass_per_block;

    grid.sync();
    accel_0();
    grid.sync();
    accel_1();
    grid.sync();
    accel_2();
    grid.sync();
    for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
        int i=i0+di+threadIdx.x;
        if(i<mn&&di+threadIdx.x<dkf.mass_per_block){
            mass &mi=x[i];
            mi.phi*=c2;
            mi.naccel*=c2;

            //ring correction
            if(mi.ringmodel){
                mass &m=mi;
                ring &mr=*m.ringmodel;
                //ring has inertia that prevents extra accelerations
                m.daccel-=(m.gaccel+m.daccel+m.naccel)*mr.GM_ratio;
                //ring has angular momentum that prevents extra angular accelerations
                fast_mpvec mGL=m.GL;
                fast_real rmGL2=1/(mGL%mGL),rmGL=sqrt(rmGL2);
                fast_mpvec ptorque=m.dtorque;
                ptorque=ptorque%mGL*rmGL2*mGL;
                fast_mpvec dtorque=m.dtorque-ptorque;
                //ptorque(parallel to GL) will not change
                //  since this part has nothing to do with the ring
                //dtorque(perpendicular to GL) will decrease
                //  due to ring's angular momentum
                dtorque*=1/(1+rmGL*mr.GL);
                m.dtorque=dtorque+ptorque;
            }
        }
    }
}

void __global__ Cuda_RungeKutta_Kernel(){
    const real *clist=(const real *)rk12_coefs;

    mass_state *x0=dkf.x0,*f=dkf.f;
    mass *x=dkf.dmlist;
    fast_real dt=dkf.dt;

    int mn=dkf.nmass;
    int i0=blockIdx.x*dkf.mass_per_block;

    for(int_t i_step=0;i_step<dkf.n_step;++i_step){
        for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
            int i=i0+di+threadIdx.x;
            if(i<mn&&di+threadIdx.x<dkf.mass_per_block){
                x0[i]=x[i];
            }
        }

        fast_real dt_k;
        for(int_t k=1;k<=25;++k){
            dt_k=0;
            for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
                int i=i0+di+threadIdx.x;
                if(i<mn&&di+threadIdx.x<dkf.mass_per_block){
                    mass_state &fi=f[mn*(k-1)+i];
                    mass &xi=x[i];
                    fi.v=xi.v;
                    fi.naccel=xi.gaccel+xi.daccel+xi.naccel;
                    fi.w=xi.w;
                    fi.dtorque=xi.dtorque;

                    if(k==25){
                        fast_mpvec j(xi.GL),&w=xi.w;
                        fast_mpvec wxj=w*j;
                        fast_real wxj2=wxj%wxj;
                        if(wxj2!=0){
                            fast_mpmat ir=xi.GI.inverse();
                            fast_real dt2=dt*dt;
                            fast_real e=w%j*(fast_real(1)/2);
                            fast_real de=(ir%w)%((ir%wxj)*j+w*wxj);
                            de*=e*dt2*dt2/(36*wxj2);
                            xi.Erot=de;
                            xi.Egrad=wxj;
                        }
                        else{
                            xi.Erot=0;
                            xi.Egrad=0;
                        }
                    }

                    xi.v=0;
                    xi.naccel=0;
                    xi.w=0;
                    xi.dtorque=0;
                }
            }

            for(int_t j=0;j<k;++j){
                const real &ckj=clist[k*(k-1)/2+j];
                const fast_real fckj=(fast_real)ckj;
                if(ckj.hi){
                    dt_k+=fckj;
                    for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
                        int i=i0+di+threadIdx.x;
                        if(i<mn&&di+threadIdx.x<dkf.mass_per_block){
                            mass_state &fi=f[mn*j+i];
                            mass &xi=x[i];
                            xi.v+=ckj*fi.v;
                            xi.naccel+=fckj*fi.naccel;
                            xi.w+=fckj*fi.w;
                            xi.dtorque+=fckj*fi.dtorque;
                        }
                    }
                }
            }
            
            fast_real t=fast_real(dkf.t_eph)+dt*(dt_k+i_step);
            for(int di=0;di<dkf.mass_per_block;di+=blockDim.x){
                int i=i0+di+threadIdx.x;
                if(i<mn&&di+threadIdx.x<dkf.mass_per_block){
                    const mass_state &xi0=x0[i];
                    mass &xi=x[i];

                    xi.r=xi0.r+real(dt)*xi.v;
                    xi.v=xi0.v+mpvec(dt*xi.naccel);
                    xi.w/=dt_k;
                    xi.s=xi0.s;
                    fast_mpvec dw=(fast_real(2)/3)*(xi.w-xi0.w);
                    xi.s+=rotation_matrix(xi.w-dw,dt*dt_k/2)%fast_mpmat(xi.s);
                    xi.s+=rotation_matrix(xi.w+dw,dt*dt_k/2)%fast_mpmat(xi.s);
                    xi.GL=xi0.GL+mpvec(dt*xi.dtorque);

                    if(k==25){
                        xi.s+=rotation_matrix(xi.Egrad,xi.Erot)%fast_mpmat(xi.s);
                    }
                    //update
                    xi.GM=xi.GM0+xi.dGM*t;
            
                    xi.exJ2=xi.dJ2*t;
                }
            }

            Cuda_accel();
        }
    }
}

void msystem::Cuda_RungeKutta12(fast_real dt,int_t n_step){
    if(n_step<=0)return;
    gpdata_t mgp;
    ringdata_t mrg;
    cuda_rungekutta_kernel_config kf;
    kf.load(mlist,mgp,mrg,dt,n_step);
    kf.t_eph=t_eph;
    hipMemcpyToSymbol(HIP_SYMBOL(dkf),&kf,sizeof(kf));
    //Cuda_Kernel<<<kf.nblocks,kf.nthreads>>>();
    hipLaunchCooperativeKernel(
        (void*)Cuda_RungeKutta_Kernel,
        dim3(kf.nblocks),
        dim3(kf.nthreads),
        nullptr,
        kf.nthreads*std::max(sizeof(maccel_1),sizeof(maccel_2))
    );
    hipDeviceSynchronize();
    kf.save(mlist,mgp,mrg);
}

void __global__ Cuda_accel_Kernel(){
    Cuda_accel();
}

void msystem::Cuda_accel(){
    gpdata_t mgp;
    ringdata_t mrg;
    cuda_rungekutta_kernel_config kf;
    kf.load(mlist,mgp,mrg,0,0);
    kf.t_eph=t_eph;
    hipMemcpyToSymbol(HIP_SYMBOL(dkf),&kf,sizeof(kf));
    //Cuda_Kernel<<<kf.nblocks,kf.nthreads>>>();
    hipLaunchCooperativeKernel(
        (void*)Cuda_RungeKutta_Kernel,
        dim3(kf.nblocks),
        dim3(kf.nthreads),
        nullptr,
        kf.nthreads*std::max(sizeof(maccel_1),sizeof(maccel_2))
    );
    hipDeviceSynchronize();
    kf.save(mlist,mgp,mrg);
}
